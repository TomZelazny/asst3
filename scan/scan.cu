#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__
void upsweep_kernel(int* input, int N, int two_d, int two_dplus1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    long long k = (long long)index * (long long)two_dplus1;

    if (k >= N) return;

    long long idx1 = k + two_d - 1;
    long long idx2 = k + two_dplus1 - 1;
    if (idx1 < N && idx2 < N) {
        input[idx2] += input[idx1];
    }
}

__global__
void downsweep_kernel(int* input, int N, int two_d, int two_dplus1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    long long k = (long long)index * (long long)two_dplus1;
    
    if (k >= N) return;
    
    long long idx1 = k + two_d - 1;
    long long idx2 = k + two_dplus1 - 1;
    if (idx1 < N && idx2 < N) {
        int t = input[idx1];
        input[idx1] = input[idx2];
        input[idx2] += t;
    }
}


// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    int rounded_length = nextPow2(N);

    if (rounded_length > N) {
        hipMemset(input + N, 0, (rounded_length - N) * sizeof(int));
    }
    
    // upsweep phase
    for (int two_d = 1; two_d < rounded_length; two_d *= 2) {
        int two_dplus1 = two_d * 2;
        // luanch one CUDA thread for each iteration for the inner loop
        long long elements_to_process = (rounded_length + two_dplus1 - 1LL) / two_dplus1;
        int num_blocks = (int)((elements_to_process + THREADS_PER_BLOCK - 1LL) / THREADS_PER_BLOCK);
        upsweep_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(input, rounded_length, two_d, two_dplus1);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error in upsweep_kernel: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
    }

    // set the last element to 0
    hipMemset(input + rounded_length - 1, 0, sizeof(int));

    // downsweep phase
    for (int two_d = rounded_length/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = two_d * 2;
        // launch one CUDA thread for each iteration for the inner loop
        long long elements_to_process = (rounded_length + two_dplus1 - 1LL) / two_dplus1;
        int num_blocks = (int)((elements_to_process + THREADS_PER_BLOCK - 1LL) / THREADS_PER_BLOCK);
        downsweep_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(input, rounded_length, two_d, two_dplus1);
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("CUDA Error in downsweep_kernel: %s\n", hipGetErrorString(err));
        // }
        hipDeviceSynchronize();
    }

    // copy the result to the output
    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__
void flag_repeats_kernel(int* input, int N, int* flags) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    flags[index] = (input[index] == input[index + 1]) ? 1 : 0;
}

__global__
void compact_kernel(int* input, int N, int* scanned_results, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1) {
        if (input[index] == input[index + 1]) {
            output[scanned_results[index]] = input[index];
        }
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int* device_flags;
    hipMalloc((void **)&device_flags, length * sizeof(int));

    // stage 1: flag repeats
    int num_threads = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int num_blocks = (num_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    flag_repeats_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_input, length - 1, device_flags);
    
    // stage 2: exclusive scan
    exclusive_scan(device_flags, length, device_flags);

    // stage 3: compact
    compact_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_input, length, device_flags, device_output);

    int total_repeats;
    hipMemcpy(&total_repeats, device_flags + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_flags);
    return total_repeats;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
